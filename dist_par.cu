
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Macro para checagem de erro das chamadas às funções do CUDA
#define checa_cuda(result) \
    if (result != hipSuccess) { \
        printf("%s\n", hipGetErrorString(result)); \
        exit(1); \
    }

char *aloca_sequencia(int n) {
    char *seq;

    seq = (char *) malloc((n + 1) * sizeof(char));
    if (seq == NULL) {
        printf("\nErro na alocação de estruturas\n");
        exit(1);
    }
    return seq;
}

__global__ void inicializa_GPU(int *a, int nLinhas, int mColunas)
{
    int i; // id GLOBAL da thread

    i = blockIdx.x * blockDim.x + threadIdx.x;

    // Inicializa as colunas da 1ª linha
    if (i < mColunas) {
        a[i] = i;
    }

    // Inicializa a 1ª coluna
    if (i < nLinhas) {
       a[i * mColunas] = i;
    }
}

// Kernel executado na GPU por todas as threads de todos os blocos
__global__ void distancia_GPU(int *a, int nLinhas, int mColunas, char *s, char *r, int *d, int deslocamento, int rodadaExt, int tamBloco)
{
    int i; // id GLOBAL da thread

    i = blockIdx.x * blockDim.x + threadIdx.x;

    int min, celulaDiagonal;

    int it = 0;  // Usado para andar com o índice de r
    
    int rodada = 0;
    int thrdIdx = threadIdx.x;

    while(rodada < 2*tamBloco - 1) {
        
        // Se (a thread estiver entre a 1ª e a última coluna do bloco  E  a 2ª e a última coluna da matriz)  E  não estiver após a última linha da matriz, nem do bloco
        if (((rodada - thrdIdx >= 0 && rodada - thrdIdx < tamBloco) && rodada - i + rodadaExt * deslocamento < mColunas-1) && i < nLinhas-1 && thrdIdx < tamBloco) {

            // Se s[i+1] e r[it+1] forem iguais, copia o valor da diagonal; senão, copia o valor da diagonal acrescido de uma unidade
            celulaDiagonal = s[i+1] == r[rodadaExt * deslocamento - blockIdx.x * deslocamento + it + 1] ?
                                            a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas - thrdIdx + rodada] :
                                            a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas - thrdIdx + rodada] + 1;

            // Mínimo entre a célula diagonal (já calculada) e a célula de cima (acrescida de uma unidade)
            min = celulaDiagonal < a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas - thrdIdx + rodada + 1] + 1 ?
                                            celulaDiagonal :
                                            a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas - thrdIdx + rodada + 1] + 1;

            // Mínimo entre a célula à esquerda e o mínimo anterior
            if (a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas + mColunas - thrdIdx + rodada] + 1 < min) {
                a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas + mColunas + 1 - thrdIdx + rodada] = a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas + mColunas - thrdIdx + rodada] + 1;
            } else {
                a[rodadaExt * deslocamento - blockIdx.x * deslocamento + i*mColunas + mColunas + 1 - thrdIdx + rodada] = min;
            }
            
            it++;
        }

        rodada++;

        // Sincronização de barreira entre todas as threads do BLOCO
        __syncthreads();
    }

    if (i == 0) {
        *d = a[nLinhas * mColunas - 1];
    }
}

// Programa principal
int main(int argc, char **argv) {
    int nLinhas,
    mColunas,
    nBytes,
    *d_a,  // Vetor (matriz de distância) da GPU (device)
    
    *d_dist,  // Variável da GPU (device) que conterá a última célula da matriz
    h_dist;   // Valor de retorno da última célula da matriz (conterá a distância)
    
    int n,  // Tamanho da sequência s
        m;  // Tamanho da sequência r

    // Sequências s/r de entrada
    char *h_s,
         *h_r,
         *d_s,
         *d_r;

    FILE *arqEntrada;  // Arquivo texto de entrada

    if(argc != 2) {
        printf("O programa foi executado com argumentos incorretos.\n");
        printf("Uso: ./dist_seq <nome arquivo entrada>\n");
        exit(1);
    }

    // Abre arquivo de entrada
    arqEntrada = fopen(argv[1], "rt");

    if (arqEntrada == NULL) {
        printf("\nArquivo texto de entrada não encontrado\n");
        exit(1);
    }

    // Lê tamanho das sequências s e r
    fscanf(arqEntrada, "%d %d", &n, &m);
    n++;
    m++;

    nLinhas = n;
    mColunas = m;
    nBytes = nLinhas * mColunas * sizeof(int);

    // Aloca vetores s e r
    h_s = aloca_sequencia(n);
    h_r = aloca_sequencia(m);

    // Lê sequências do arquivo de entrada
    h_s[0] = ' ';
    h_r[0] = ' ';
    fscanf(arqEntrada, "%s", &(h_s[1]));
    fscanf(arqEntrada, "%s", &(h_r[1]));

    // Fecha arquivo de entrada
    fclose(arqEntrada);

    
    /* Alocação de memória e checagem de erro */

    // Aloca vetor (matriz de distância) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_a, nBytes));

    // Aloca variável (distância) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_dist, sizeof(int)));
     
    // Aloca vetor (sequência r) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_r, m*sizeof(char)));

    // Aloca vetor (sequência s) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_s, n*sizeof(char)));
    
    
    hipEvent_t d_ini, d_fim;
    hipEventCreate(&d_ini);
    hipEventCreate(&d_fim);
    hipEventRecord(d_ini, 0);
    
    // Máximo entre a quantidade de linhas e de colunas
    int n_threads_bloco = n > m ? n : m;
    
    // Se a quantidade pega for maior que 1024, pega o valor 1024, pois esta é a quantidade máxima de threads que cabem em um bloco
    if (n_threads_bloco > 1024) {
        n_threads_bloco = 1024;
    }
    
    // Determina nBlocos em função de mColunas e n_threads_bloco,
    // ou seja, calcula a quantidade de blocos necessária para cobrir todas as linhas
    int nBlocos = (mColunas + n_threads_bloco - 1) / n_threads_bloco;
    
    inicializa_GPU<<<nBlocos, n_threads_bloco>>>(d_a, nLinhas, mColunas);

    // Copia a sequência s do host para a GPU e checa se houve erro
    checa_cuda(hipMemcpy(d_s, h_s, n*sizeof(char), hipMemcpyHostToDevice));

    // Copia a sequência r do host para a GPU e checa se houve erro
    checa_cuda(hipMemcpy(d_r, h_r, m*sizeof(char), hipMemcpyHostToDevice));
    
    // Host espera GPU terminar de executar
    hipDeviceSynchronize();
    
    // Define o tamanho do bloco baseado na quantidade de colunas
    int tamBloco = (mColunas-1) <= 1024 ? mColunas-1 : 1024;
    
    int deslocamento = tamBloco;
    nBlocos = 1;
    
    int rodadaExt = 0;
    
    // Total de repetições necessárias para um bloco percorrer a matriz da esquerda para a direita.
    //   Ex: em uma matriz com 3000 colunas, 1 bloco de tamanho 1024 precisará ser chamado 3 vezes para percorrer a matriz da esq p/ dir
    int blocosLinha = (mColunas-1) % tamBloco == 0 ? (mColunas-1) / tamBloco : (mColunas-1) / tamBloco + 1;
    
    // Total de repetições necessárias para o último bloco começar a processar a última porção da matriz.
    //   Ex: em uma matriz com 2000 linhas, serão necessárias 2 rodadas para um bloco de tamanho 1024 iniciar o processamento da última porção da matriz
    int blocosColuna = (nLinhas-1) % tamBloco == 0 ? (nLinhas-1) / tamBloco : (nLinhas-1) / tamBloco + 1;
    
    // Total de repetições para todos os blocos percorrerem a matriz
    int repeticoes = blocosLinha + blocosColuna;
    
    int linhasRestantes = nLinhas;
    
    while (repeticoes-- > 0) {
 
        // Calcula a distância de edição na GPU
        distancia_GPU<<<nBlocos, n_threads_bloco>>>(d_a, nLinhas, mColunas, d_s, d_r, d_dist, deslocamento, rodadaExt, tamBloco);
        
        rodadaExt++;
        
        linhasRestantes = linhasRestantes - n_threads_bloco;
        if (linhasRestantes > 0)
            nBlocos++;
        
        // Host espera GPU terminar de executar
        hipDeviceSynchronize();
    }

    // Copia a distância (última célula da matriz) para o host
    checa_cuda(hipMemcpy(&h_dist, d_dist, sizeof(int), hipMemcpyDeviceToHost));

    hipEventRecord(d_fim, 0);
    hipEventSynchronize(d_fim);
    float d_tempo;      // Tempo de execução na GPU em milissegundos
    hipEventElapsedTime(&d_tempo, d_ini, d_fim);
    hipEventDestroy(d_ini);
    hipEventDestroy(d_fim);

    printf("%d\n", h_dist);
    printf("%.2f\n", d_tempo);

    // Libera vetor (matriz de distância) da memória global da GPU
    hipFree(d_a);

    // Libera vetores da memória global da GPU
    hipFree(d_s);
    hipFree(d_r);

    // Libera vetores da memória do host
    free(h_s);
    free(h_r);

    // Libera variável da memória global da GPU
    hipFree(d_dist);

    return 0;
}